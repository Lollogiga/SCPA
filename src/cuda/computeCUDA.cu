#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "../include/computeCUDA.cuh"

// __global__ void cudaKernel(int *d_data, int N) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < N) {
//         d_data[idx] += 1;
//     }
// }

extern "C" int computeCUDA() {
    // const int size = 10;
    // int h_data[size] = {0}; // Array sulla CPU
    // int *d_data;
    //
    // // Alloca memoria sulla GPU
    // if (hipMalloc((void **)&d_data, size * sizeof(int)) != hipSuccess) {
    //     printf("Errore di allocazione memoria GPU!\n");
    //     return;
    // }
    //
    // // Azzeramento della memoria sulla GPU (opzionale ma consigliato)
    // hipMemset(d_data, 0, size * sizeof(int));
    //
    // // Copia dati dalla CPU alla GPU
    // if (hipMemcpy(d_data, h_data, size * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
    //     printf("Errore nella copia CPU -> GPU!\n");
    //     hipFree(d_data);
    //     return;
    // }
    //
    // // Lancio del kernel
    // cudaKernel<<<1, 10>>>(d_data, size);
    //
    // // Controllo errori CUDA dopo il lancio del kernel
    // hipError_t err = hipGetLastError();
    // if (err != hipSuccess) {
    //     printf("Errore nel lancio del kernel: %s\n", hipGetErrorString(err));
    //     hipFree(d_data);
    //     return;
    // }
    //
    // // Copia risultati dalla GPU alla CPU
    // if (hipMemcpy(h_data, d_data, size * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
    //     printf("Errore nella copia GPU -> CPU!\n");
    //     hipFree(d_data);
    //     return;
    // }
    //
    // // Stampa il risultato
    // printf("Risultati CUDA: ");
    // for (int i = 0; i < size; i++) {
    //     printf("%d ", h_data[i]);
    // }
    // printf("\n");
    //
    // // Libera memoria GPU
    // hipFree(d_data);

    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    if (err != hipSuccess) {
        printf("Errore nel recuperare il numero di dispositivi CUDA: %s\n", hipGetErrorString(err));
        return -1;
    }

    for (int i = 0; i < deviceCount; ++i) {
        struct hipDeviceProp_t deviceProp;
        err = hipGetDeviceProperties(&deviceProp, i);
        if (err != hipSuccess) {
            printf("Errore nel recuperare le proprietà del dispositivo %d: %s\n", i, hipGetErrorString(err));
            continue;
        }

        printf("Device %d: %s\n", i, deviceProp.name);
        printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    }

    return 0;
}
