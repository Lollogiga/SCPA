#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#include "../include/openmp/Serial.h"
#include "../include/cuda/Serial.cuh"
#include "../include/cuda/CSR.cuh"
#include "../include/cuda/HLL.cuh"
#include "../include/cuda/Utils.cuh"
#include "../include/checkResultVector.h"
#include "../include/mtxStructs.h"
#include "../include/flops.h"

/*
 * --- NOTE SULLA SCHEDA VIDEO SUL SERVER DI DIPARTIMENTO ---
 * Nome GPU: Quadro RTX 5000
 * Max threads per block: 1024
 * Warp size: 32
 * Max threads per multiprocessore: 1024
 * Numero di multiprocessori: 48
 * Max blocchi per griglia: 2147483647
 * Memoria condivisa per blocco: 49152 bytes
 */

int csr_product(CSRMatrix *h_csr, ResultVector *serial) {
    hipError_t cuda_error;
    int int_err;
    
    float elapsedTime;
    double check;
    
    CSRMatrix *d_csr = uploadCSRToDevice(h_csr);
    if (d_csr == nullptr) {
        freeCSRDevice(d_csr);
        return -1;
    }

    int threadsPerBlock = BLOCK_SIZE;
    int warpsPerBlock = threadsPerBlock / WARP_SIZE;
    int blocksPerGrid = (h_csr->M + warpsPerBlock - 1) / warpsPerBlock;

    MatVal *h_x = create_vector(h_csr->N);
    if (h_x == nullptr) {
        printf("\033[31mcsr_product - create_vector h_x->N failed\033[0m\n");

        freeCSRDevice(d_csr);
        return -1;
    }

    MatVal* d_x;
    cuda_error = hipMallocManaged(&d_x, h_csr->N * sizeof(MatVal));
    if (cuda_error != hipSuccess) {
        printf("\033[31mcsr_product - hipMallocManaged h_csr->N failed: %s\033[0m\n", hipGetErrorString(cuda_error));

        freeCSRDevice(d_csr);
        free_vector(h_x);
        return -1;
    }
    cuda_error = hipMemcpy(d_x, h_x, h_csr->N * sizeof(MatVal), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        printf("\033[31mcsr_product - hipMemcpy h_x failed: %s\033[0m\n", hipGetErrorString(cuda_error));

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        return -1;
    }
    cuda_error = hipMemAdvise(d_x, h_csr->N * sizeof(MatVal), hipMemAdviseSetReadMostly, 0);
    if (cuda_error != hipSuccess) {
        printf("\033[31mcsr_product - hipMemAdvise SetReadMostly failed: %s\033[0m\n", hipGetErrorString(cuda_error));

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        return -1;
    }
    cuda_error = hipMemAdvise(d_x, h_csr->N * sizeof(MatVal), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    if (cuda_error != hipSuccess) {
        printf("\033[31mcsr_product - hipMemAdvise SetPreferredLocation failed: %s\033[0m\n", hipGetErrorString(cuda_error));

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        return -1;
    }

    ResultVector *h_result_vector = create_result_vector(h_csr->M);
    if (h_result_vector == nullptr) {
        printf("\033[31mcsr_product - create_result_vector h_csr->M failed\033[0m\n");

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        return -1;
    }
    ResultVector *d_result_vector = uploadResultVectorToDevice(h_result_vector);
    if (d_result_vector == nullptr) {
        printf("\033[31mcsr_product - uploadResultVectorToDevice h_result_vector failed\033[0m\n");

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        return -1;
    }

    MatVal* d_y;
    cuda_error = hipMalloc(&d_y, h_csr->M * sizeof(MatVal));
    if (cuda_error != hipSuccess) {
        printf("\033[31mcsr_product - hipMalloc d_y failed: %s\033[0m\n", hipGetErrorString(cuda_error));

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        return -1;
    }

    CUDA_EVENT_CREATE(start, stop)

    // SOL SERIAL
    CUDA_EVENT_START(start)
    spmv_csr_serial<<<1, 1>>>(d_csr, d_x, d_result_vector);
    cuda_error = hipGetLastError();
    CUDA_EVENT_STOP(stop)
    CUDA_EVENT_ELAPSED(start, stop, elapsedTime)
    if (cuda_error) {
        printf("\033[31mcsr_product - spmv_csr_serial kernel failed: %s\033[0m\n", hipGetErrorString(cuda_error));

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        CUDA_EVENT_DESTROY(start, stop)
        return -1;
    }
    printf("CudaSerial: Flops: %f\n", computeFlops(h_csr->NZ, elapsedTime));
    int_err = downloadResultVectorToHost(h_result_vector, d_result_vector);
    if (int_err != 0) {
        printf("\033[31mcsr_product - downloadResultVectorToHost spmv_csr_serial failed\033[0m\n");

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        CUDA_EVENT_DESTROY(start, stop)
        return -1;
    }
    int_err = checkResultVector(serial, h_result_vector);
    if (int_err) {
        printf("\033[31mcsr_product - checkResultVector spmv_csr_serial failed\033[0m\n");

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        CUDA_EVENT_DESTROY(start, stop)
        return -1;
    }
    freeResultVectorFromDevice(d_result_vector);

    // SOL 1
    for (int i = 0; i < h_csr->M; i++) h_result_vector->val[i] = 0;
    d_result_vector = uploadResultVectorToDevice(h_result_vector);
    if (d_result_vector == nullptr) {
        printf("\033[31mcsr_product - uploadResultVectorToDevice h_result_vector failed\033[0m\n");

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        CUDA_EVENT_DESTROY(start, stop)
        return -1;
    }

    CUDA_EVENT_START(start)
    csr_cudaProduct_sol1<<<blocksPerGrid, threadsPerBlock>>>(d_csr, d_x, d_result_vector);
    CUDA_EVENT_STOP(stop)
    cuda_error = hipGetLastError();
    CUDA_EVENT_ELAPSED(start, stop, elapsedTime)
    if (cuda_error) {
        printf("\033[31mcsr_product - csr_cudaProduct_sol1 kernel failed: %s\033[0m\n", hipGetErrorString(cuda_error));

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        CUDA_EVENT_DESTROY(start, stop)
        return -1;
    }
    printf("csr_cudaProduct_sol1: Flops: %f\n", computeFlops(h_csr->NZ, elapsedTime));
    int_err = downloadResultVectorToHost(h_result_vector, d_result_vector);
    if (int_err != 0) {
        printf("\033[31mcsr_product - downloadResultVectorToHost csr_cudaProduct_sol1 failed\033[0m\n");

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        CUDA_EVENT_DESTROY(start, stop)
        return -1;
    }
    int_err = checkResultVector(serial, h_result_vector);
    if (int_err) {
        printf("\033[31mcsr_product - checkResultVector csr_cudaProduct_sol1 failed\033[0m\n");

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        CUDA_EVENT_DESTROY(start, stop)
        return -1;
    }
    freeResultVectorFromDevice(d_result_vector);

    // SOL 2
    for (int i = 0; i < h_csr->M; i++) h_result_vector->val[i] = 0;
    d_result_vector = uploadResultVectorToDevice(h_result_vector);
    if (d_result_vector == nullptr) {
        printf("\033[31mcsr_product - uploadResultVectorToDevice h_result_vector failed\033[0m\n");

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        CUDA_EVENT_DESTROY(start, stop)
        return -1;
    }

    MatVal* d_product;
    cuda_error = hipMalloc(&d_product, h_csr->NZ * sizeof(MatVal));
    if (cuda_error != hipSuccess) {
        printf("\033[31mcsr_product - hipMalloc d_product failed: %s\033[0m\n", hipGetErrorString(cuda_error));

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        CUDA_EVENT_DESTROY(start, stop)
        return -1;
    }

    blocksPerGrid = (h_csr->NZ + warpsPerBlock - 1) / warpsPerBlock;

    cuda_error = hipGetLastError();
    if (cuda_error) {
        printf("\033[31mMy supposition: %s\033[0m\n", hipGetErrorString(cuda_error));

        return -1;
    }

    CUDA_EVENT_START(start)
    csr_cudaProduct_sol2_product<<<blocksPerGrid, threadsPerBlock>>>(d_csr, d_product, d_x);
    hipDeviceSynchronize();

    blocksPerGrid = (h_csr->M + warpsPerBlock - 1) / warpsPerBlock;
    csr_cudaProduct_sol2_reduce<<<blocksPerGrid, threadsPerBlock>>>(d_csr, d_product, d_result_vector);
    CUDA_EVENT_STOP(stop)
    cuda_error = hipGetLastError();
    CUDA_EVENT_ELAPSED(start, stop, elapsedTime)
    if (cuda_error) {
        printf("\033[31mcsr_product - csr_cudaProduct_sol1 kernel failed: %s\033[0m\n", hipGetErrorString(cuda_error));

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        CUDA_EVENT_DESTROY(start, stop)
        return -1;
    }
    printf("csr_cudaProduct_sol2: Flops: %f\n", computeFlops(h_csr->NZ, elapsedTime));
    int_err = downloadResultVectorToHost(h_result_vector, d_result_vector);
    if (int_err != 0) {
        printf("\033[31mcsr_product - downloadResultVectorToHost csr_cudaProduct_sol2 failed\033[0m\n");

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        CUDA_EVENT_DESTROY(start, stop)
        return -1;
    }
    int_err = checkResultVector(serial, h_result_vector);
    if (int_err) {
        printf("\033[31mcsr_product - checkResultVector csr_cudaProduct_sol2 failed\033[0m\n");

        freeCSRDevice(d_csr);
        free_vector(h_x);
        hipFree(d_x);
        free_result_vector(h_result_vector);
        freeResultVectorFromDevice(d_result_vector);
        CUDA_EVENT_DESTROY(start, stop)
        return -1;
    }
    freeResultVectorFromDevice(d_result_vector);
    hipFree(d_product);

    // SOL 3
    for (int i = 0; i < h_csr->M; i++) h_result_vector->val[i] = 0;
    d_result_vector = uploadResultVectorToDevice(h_result_vector);

    CUDA_EVENT_START(start)
    csr_cudaProduct_sol3<<<blocksPerGrid, threadsPerBlock>>>(d_csr, d_x, d_result_vector);
    CUDA_EVENT_STOP(stop)
    CUDA_EVENT_ELAPSED(start, stop, elapsedTime)
    printf("CudaSol2: Flops: %f\n", computeFlops(h_csr->NZ, elapsedTime));
    downloadResultVectorToHost(h_result_vector, d_result_vector);
    check = checkResultVector(serial, h_result_vector);
    if (check) {
        printf("check = %.0f\n", check);
        perror("\033[31mError checkResultVector in CudaSol2\033[0m\n");

        return -1;
    }
    freeResultVectorFromDevice(d_result_vector);

    freeCSRDevice(d_csr);
    hipFree(d_x);
    hipFree(d_y);

    CUDA_EVENT_DESTROY(start, stop)

    free(h_x);

    return 0;
}

extern "C" int computeCUDA(CSRMatrix *csr, HLLMatrix *hll, HLLMatrixAligned *hllAligned, int num_threads) {
    MatVal *vector = create_vector(csr->N);
    if (vector == nullptr) return -1;

    ResultVector *serial = csr_serialProduct(csr, vector);

    csr_product(csr, serial);
    // hll_CUDA_product(hll, serial);

    int sharedMemPerBlock;
    hipDeviceGetAttribute(&sharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);


    // hipDeviceProp_t prop;
    // hipGetDeviceProperties(&prop, 0);

    // printf("Nome GPU: %s\n", prop.name);
    // printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    // printf("Warp size: %d\n", prop.warpSize);
    // printf("Max threads per multiprocessore: %d\n", prop.maxThreadsPerMultiProcessor);
    // printf("Numero di multiprocessori: %d\n", prop.multiProcessorCount);
    // printf("Max blocchi per griglia: %d\n", prop.maxGridSize[0]);
    // printf("Memoria condivisa per blocco: %zu bytes\n", prop.sharedMemPerBlock);

    return 0;
}
