#include "../include/cuda/Utils.cuh"

CSRMatrix* uploadCSRToDevice(const CSRMatrix *h_csr) {
    // 1. Crea struttura temporanea su host
    CSRMatrix d_csr;
    d_csr.M = h_csr->M;
    d_csr.N = h_csr->N;
    d_csr.NZ = h_csr->NZ;

    // 2. Alloca array su device e copia i contenuti
    hipMalloc((void**)&d_csr.IRP, (h_csr->M + 1) * sizeof(MatT));
    hipMemcpy(d_csr.IRP, h_csr->IRP, (h_csr->M + 1) * sizeof(MatT), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_csr.JA, h_csr->NZ * sizeof(MatT));
    hipMemcpy(d_csr.JA, h_csr->JA, h_csr->NZ * sizeof(MatT), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_csr.AS, h_csr->NZ * sizeof(MatVal));
    hipMemcpy(d_csr.AS, h_csr->AS, h_csr->NZ * sizeof(MatVal), hipMemcpyHostToDevice);

    // 3. Alloca CSRMatrix su device
    CSRMatrix *d_csr_ptr;
    hipMalloc((void**)&d_csr_ptr, sizeof(CSRMatrix));

    // 4. Copia la struttura su device
    hipMemcpy(d_csr_ptr, &d_csr, sizeof(CSRMatrix), hipMemcpyHostToDevice);

    return d_csr_ptr;
}

ResultVector* uploadResultVectorToDevice(const ResultVector *h_vec) {
    // 1. Crea struttura temporanea su host
    ResultVector d_vec;
    d_vec.len_vector = h_vec->len_vector;

    // 2. Alloca e copia l'array dinamico su device
    hipMalloc((void**)&d_vec.val, h_vec->len_vector * sizeof(MatVal));
    hipMemcpy(d_vec.val, h_vec->val, h_vec->len_vector * sizeof(MatVal), hipMemcpyHostToDevice);

    // 3. Alloca la struttura ResultVector su device
    ResultVector *d_vec_ptr;
    hipMalloc((void**)&d_vec_ptr, sizeof(ResultVector));

    // 4. Copia la struttura (con puntatore già device) sul device
    hipMemcpy(d_vec_ptr, &d_vec, sizeof(ResultVector), hipMemcpyHostToDevice);

    return d_vec_ptr;
}

void downloadResultVectorToHost(ResultVector *hostResultVector, const ResultVector *deviceResultVector) {
    hipMemcpy(hostResultVector, deviceResultVector, sizeof(ResultVector), hipMemcpyDeviceToHost);

    MatVal *device_val_ptr = hostResultVector->val;

    hostResultVector->val = (MatVal*)malloc(hostResultVector->len_vector * sizeof(MatVal));

    hipMemcpy(hostResultVector->val, device_val_ptr, hostResultVector->len_vector * sizeof(MatVal), hipMemcpyDeviceToHost);
}

void freeCSRDevice(CSRMatrix *d_csr_ptr) {
    CSRMatrix h_temp;
    hipMemcpy(&h_temp, d_csr_ptr, sizeof(CSRMatrix), hipMemcpyDeviceToHost);

    hipFree(h_temp.IRP);
    hipFree(h_temp.JA);
    hipFree(h_temp.AS);
    hipFree(d_csr_ptr);
}

void freeResultVectorFromDevice(ResultVector *d_result_vector) {
    // 1. Copia la struttura dal device all'host per accedere ai campi
    ResultVector h_temp;
    hipMemcpy(&h_temp, d_result_vector, sizeof(ResultVector), hipMemcpyDeviceToHost);

    // 2. Libera il campo val (array device)
    hipFree(h_temp.val);

    // 3. Libera la struttura vera e propria (device)
    hipFree(d_result_vector);
}

