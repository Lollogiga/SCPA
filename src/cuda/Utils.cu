#include "../include/cuda/Utils.cuh"

#include <stdio.h>

CSRMatrix* uploadCSRToDevice(const CSRMatrix *h_csr) {
    hipError_t err;

    CSRMatrix d_csr;
    d_csr.M = h_csr->M;
    d_csr.N = h_csr->N;
    d_csr.NZ = h_csr->NZ;

    err = hipMalloc((void**)&d_csr.IRP, (h_csr->M + 1) * sizeof(MatT));
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMalloc IRP failed: %s\033[0m\n", hipGetErrorString(err));
        return nullptr;
    }
    err = hipMemcpy(d_csr.IRP, h_csr->IRP, (h_csr->M + 1) * sizeof(MatT), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMemcpy IRP failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        return nullptr;
    }

    err = hipMalloc((void**)&d_csr.JA, h_csr->NZ * sizeof(MatT));
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMalloc JA failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        return nullptr;
    }
    err = hipMemcpy(d_csr.JA, h_csr->JA, h_csr->NZ * sizeof(MatT), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMemcpy JA failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        hipFree(d_csr.JA);
        return nullptr;
    }

    err = hipMalloc((void**)&d_csr.AS, h_csr->NZ * sizeof(MatVal));
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMalloc AS failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        hipFree(d_csr.JA);
        return nullptr;
    }
    err = hipMemcpy(d_csr.AS, h_csr->AS, h_csr->NZ * sizeof(MatVal), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMemcpy AS failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        hipFree(d_csr.JA);
        hipFree(d_csr.AS);
        return nullptr;
    }

    CSRMatrix *d_csr_ptr;
    err = hipMalloc((void**)&d_csr_ptr, sizeof(CSRMatrix));
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMalloc CSRMatrix failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        hipFree(d_csr.JA);
        hipFree(d_csr.AS);
        return nullptr;
    }
    err = hipMemcpy(d_csr_ptr, &d_csr, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMemcpy CSRMatrix failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        hipFree(d_csr.JA);
        hipFree(d_csr.AS);
        hipFree(d_csr_ptr);
        return nullptr;
    }

    return d_csr_ptr;
}
int freeCSRDevice(CSRMatrix *d_csr_ptr) {
    hipError_t err;

    CSRMatrix h_temp;
    err = hipMemcpy(&h_temp, d_csr_ptr, sizeof(CSRMatrix), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("\033[31mfreeCSRDevice - hipMemcpy CSRMatrix failed: %s\033[0m\n", hipGetErrorString(err));

        return -1;
    }

    hipFree(h_temp.IRP);
    hipFree(h_temp.JA);
    hipFree(h_temp.AS);
    hipFree(d_csr_ptr);

    return 0;
}

HLLMatrix* uploadHLLToDevice(const HLLMatrix* h_hll) {
    HLLMatrix* d_hll;
    hipMalloc(&d_hll, sizeof(HLLMatrix));

    // Copia metadati base
    hipMemcpy(d_hll, h_hll, sizeof(HLLMatrix), hipMemcpyHostToDevice);

    // Alloca array di puntatori ai blocchi su device
    ELLPACKMatrix** d_blocks;
    hipMalloc(&d_blocks, h_hll->numBlocks * sizeof(ELLPACKMatrix*));

    for (MatT i = 0; i < h_hll->numBlocks; i++) {
        ELLPACKMatrix* h_block = h_hll->blocks[i];
        ELLPACKMatrix* d_block;
        hipMalloc(&d_block, sizeof(ELLPACKMatrix));

        // Copia campi non-puntatore
        hipMemcpy(d_block, h_block, sizeof(ELLPACKMatrix), hipMemcpyHostToDevice);

        // Gestione JA
        MatT** d_JA;
        hipMalloc(&d_JA, h_block->M * sizeof(MatT*));
        for (MatT j = 0; j < h_block->M; j++) {
            MatT* d_row;
            hipMalloc(&d_row, h_block->MAXNZ * sizeof(MatT));
            hipMemcpy(d_row, h_block->JA[j], h_block->MAXNZ * sizeof(MatT), hipMemcpyHostToDevice);
            hipMemcpy(&d_JA[j], &d_row, sizeof(MatT*), hipMemcpyHostToDevice);
        }
        hipMemcpy(&d_block->JA, &d_JA, sizeof(MatT**), hipMemcpyHostToDevice);

        // Gestione AS (analogo a JA)
        MatVal** d_AS;
        hipMalloc(&d_AS, h_block->M * sizeof(MatVal*));
        for (MatT j = 0; j < h_block->M; j++) {
            MatVal* d_row;
            hipMalloc(&d_row, h_block->MAXNZ * sizeof(MatVal));
            hipMemcpy(d_row, h_block->AS[j], h_block->MAXNZ * sizeof(MatVal), hipMemcpyHostToDevice);
            hipMemcpy(&d_AS[j], &d_row, sizeof(MatVal*), hipMemcpyHostToDevice);
        }
        hipMemcpy(&d_block->AS, &d_AS, sizeof(MatVal**), hipMemcpyHostToDevice);

        // Aggiorna array blocchi
        hipMemcpy(&d_blocks[i], &d_block, sizeof(ELLPACKMatrix*), hipMemcpyHostToDevice);
    }

    hipMemcpy(&d_hll->blocks, &d_blocks, sizeof(ELLPACKMatrix**), hipMemcpyHostToDevice);
    return d_hll;
}

void freeHLLDevice(HLLMatrix* d_hll) {
    ELLPACKMatrix** d_blocks;
    hipMemcpy(&d_blocks, &d_hll->blocks, sizeof(ELLPACKMatrix**), hipMemcpyDeviceToHost);

    MatT numBlocks;
    hipMemcpy(&numBlocks, &d_hll->numBlocks, sizeof(MatT), hipMemcpyDeviceToHost);

    for (MatT i = 0; i < numBlocks; i++) {
        ELLPACKMatrix* d_block;
        hipMemcpy(&d_block, &d_blocks[i], sizeof(ELLPACKMatrix*), hipMemcpyDeviceToHost);

        MatT M;
        hipMemcpy(&M, &d_block->M, sizeof(MatT), hipMemcpyDeviceToHost);

        // Dealloca JA
        MatT** d_JA;
        hipMemcpy(&d_JA, &d_block->JA, sizeof(MatT**), hipMemcpyDeviceToHost);
        for (MatT j = 0; j < M; j++) {
            MatT* d_row;
            hipMemcpy(&d_row, &d_JA[j], sizeof(MatT*), hipMemcpyDeviceToHost);
            hipFree(d_row);
        }
        hipFree(d_JA);

        // Dealloca AS (analogo a JA)
        MatVal** d_AS;
        hipMemcpy(&d_AS, &d_block->AS, sizeof(MatVal**), hipMemcpyDeviceToHost);
        for (MatT j = 0; j < M; j++) {
            MatVal* d_row;
            hipMemcpy(&d_row, &d_AS[j], sizeof(MatVal*), hipMemcpyDeviceToHost);
            hipFree(d_row);
        }
        hipFree(d_AS);

        hipFree(d_block);
    }

    hipFree(d_blocks);
    hipFree(d_hll);
}




ResultVector* uploadResultVectorToDevice(const ResultVector *h_vec) {
    hipError_t err;

    ResultVector d_vec;
    d_vec.len_vector = h_vec->len_vector;

    err = hipMalloc((void**)&d_vec.val, h_vec->len_vector * sizeof(MatVal));
    if (err != hipSuccess) {
        printf("\033[31muploadResultVectorToDevice - hipMalloc val failed: %s\033[0m\n", hipGetErrorString(err));
        return nullptr;
    }
    err = hipMemcpy(d_vec.val, h_vec->val, h_vec->len_vector * sizeof(MatVal), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\033[31muploadResultVectorToDevice - hipMemcpy val failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_vec.val);
        return nullptr;
    }

    ResultVector *d_vec_ptr;
    err = hipMalloc((void**)&d_vec_ptr, sizeof(ResultVector));
    if (err != hipSuccess) {
        printf("\033[31muploadResultVectorToDevice - hipMalloc ResultVector failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_vec.val);
        return nullptr;
    }
    err = hipMemcpy(d_vec_ptr, &d_vec, sizeof(ResultVector), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\033[31muploadResultVectorToDevice - hipMemcpy ResultVector failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_vec.val);
        hipFree(d_vec_ptr);
        return nullptr;
    }

    return d_vec_ptr;
}



int freeResultVectorFromDevice(ResultVector *d_result_vector) {
    hipError_t err;

    ResultVector h_temp;
    err = hipMemcpy(&h_temp, d_result_vector, sizeof(ResultVector), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("\033[31mfreeResultVectorFromDevice - hipMemcpy ResultVector failed: %s\033[0m\n", hipGetErrorString(err));

        return -1;
    }

    hipFree(h_temp.val);
    hipFree(d_result_vector);

    return 0;
}

int downloadResultVectorToHost(ResultVector *hostResultVector, const ResultVector *deviceResultVector) {
    hipError_t err;

    err = hipMemcpy(hostResultVector, deviceResultVector, sizeof(ResultVector), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("\033[31mdownloadResultVectorToHost - hipMemcpy ResultVector failed: %s\033[0m\n", hipGetErrorString(err));

        printf("hostResultVector: %p, deviceResultVector: %p\n", hostResultVector, deviceResultVector);

        return -1;
    }

    MatVal *device_val_ptr = hostResultVector->val;

    hostResultVector->val = (MatVal*)malloc(hostResultVector->len_vector * sizeof(MatVal));
    if (hostResultVector->val == NULL) {
        printf("\033[31mdownloadResultVectorToHost - Memory allocation for val failed\033[0m\n");
        return -1;
    }

    err = hipMemcpy(hostResultVector->val, device_val_ptr, hostResultVector->len_vector * sizeof(MatVal), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("\033[31mdownloadResultVectorToHost - hipMemcpy val failed: %s\033[0m\n", hipGetErrorString(err));

        return -1;
    }

    return 0;
}