#include "../include/cuda/Utils.cuh"

#include <stdio.h>

CSRMatrix* uploadCSRToDevice(const CSRMatrix *h_csr) {
    hipError_t err;

    CSRMatrix d_csr;
    d_csr.M = h_csr->M;
    d_csr.N = h_csr->N;
    d_csr.NZ = h_csr->NZ;

    err = hipMalloc((void**)&d_csr.IRP, (h_csr->M + 1) * sizeof(MatT));
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMalloc IRP failed: %s\033[0m\n", hipGetErrorString(err));
        return nullptr;
    }
    err = hipMemcpy(d_csr.IRP, h_csr->IRP, (h_csr->M + 1) * sizeof(MatT), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMemcpy IRP failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        return nullptr;
    }

    err = hipMalloc((void**)&d_csr.JA, h_csr->NZ * sizeof(MatT));
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMalloc JA failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        return nullptr;
    }
    err = hipMemcpy(d_csr.JA, h_csr->JA, h_csr->NZ * sizeof(MatT), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMemcpy JA failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        hipFree(d_csr.JA);
        return nullptr;
    }

    err = hipMalloc((void**)&d_csr.AS, h_csr->NZ * sizeof(MatVal));
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMalloc AS failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        hipFree(d_csr.JA);
        return nullptr;
    }
    err = hipMemcpy(d_csr.AS, h_csr->AS, h_csr->NZ * sizeof(MatVal), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMemcpy AS failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        hipFree(d_csr.JA);
        hipFree(d_csr.AS);
        return nullptr;
    }

    CSRMatrix *d_csr_ptr;
    err = hipMalloc((void**)&d_csr_ptr, sizeof(CSRMatrix));
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMalloc CSRMatrix failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        hipFree(d_csr.JA);
        hipFree(d_csr.AS);
        return nullptr;
    }
    err = hipMemcpy(d_csr_ptr, &d_csr, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\033[31muploadCSRToDevice - hipMemcpy CSRMatrix failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_csr.IRP);
        hipFree(d_csr.JA);
        hipFree(d_csr.AS);
        hipFree(d_csr_ptr);
        return nullptr;
    }

    return d_csr_ptr;
}

int freeCSRDevice(CSRMatrix *d_csr_ptr) {
    hipError_t err;

    CSRMatrix h_temp;
    err = hipMemcpy(&h_temp, d_csr_ptr, sizeof(CSRMatrix), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("\033[31mfreeCSRDevice - hipMemcpy CSRMatrix failed: %s\033[0m\n", hipGetErrorString(err));

        return -1;
    }

    hipFree(h_temp.IRP);
    hipFree(h_temp.JA);
    hipFree(h_temp.AS);
    hipFree(d_csr_ptr);

    return 0;
}

ResultVector* uploadResultVectorToDevice(const ResultVector *h_vec) {
    hipError_t err;

    ResultVector d_vec;
    d_vec.len_vector = h_vec->len_vector;

    err = hipMalloc((void**)&d_vec.val, h_vec->len_vector * sizeof(MatVal));
    if (err != hipSuccess) {
        printf("\033[31muploadResultVectorToDevice - hipMalloc val failed: %s\033[0m\n", hipGetErrorString(err));
        return nullptr;
    }
    err = hipMemcpy(d_vec.val, h_vec->val, h_vec->len_vector * sizeof(MatVal), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\033[31muploadResultVectorToDevice - hipMemcpy val failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_vec.val);
        return nullptr;
    }

    ResultVector *d_vec_ptr;
    err = hipMalloc((void**)&d_vec_ptr, sizeof(ResultVector));
    if (err != hipSuccess) {
        printf("\033[31muploadResultVectorToDevice - hipMalloc ResultVector failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_vec.val);
        return nullptr;
    }
    err = hipMemcpy(d_vec_ptr, &d_vec, sizeof(ResultVector), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("\033[31muploadResultVectorToDevice - hipMemcpy ResultVector failed: %s\033[0m\n", hipGetErrorString(err));

        hipFree(d_vec.val);
        hipFree(d_vec_ptr);
        return nullptr;
    }

    return d_vec_ptr;
}

int freeResultVectorFromDevice(ResultVector *d_result_vector) {
    hipError_t err;

    ResultVector h_temp;
    err = hipMemcpy(&h_temp, d_result_vector, sizeof(ResultVector), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("\033[31mfreeResultVectorFromDevice - hipMemcpy ResultVector failed: %s\033[0m\n", hipGetErrorString(err));

        return -1;
    }

    hipFree(h_temp.val);
    hipFree(d_result_vector);

    return 0;
}

int downloadResultVectorToHost(ResultVector *hostResultVector, const ResultVector *deviceResultVector) {
    hipError_t err;

    err = hipMemcpy(hostResultVector, deviceResultVector, sizeof(ResultVector), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("\033[31mdownloadResultVectorToHost - hipMemcpy ResultVector failed: %s\033[0m\n", hipGetErrorString(err));

        printf("hostResultVector: %p, deviceResultVector: %p\n", hostResultVector, deviceResultVector);

        return -1;
    }

    MatVal *device_val_ptr = hostResultVector->val;

    hostResultVector->val = (MatVal*)malloc(hostResultVector->len_vector * sizeof(MatVal));
    if (hostResultVector->val == NULL) {
        printf("\033[31mdownloadResultVectorToHost - Memory allocation for val failed\033[0m\n");
        return -1;
    }

    err = hipMemcpy(hostResultVector->val, device_val_ptr, hostResultVector->len_vector * sizeof(MatVal), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("\033[31mdownloadResultVectorToHost - hipMemcpy val failed: %s\033[0m\n", hipGetErrorString(err));

        return -1;
    }

    return 0;
}