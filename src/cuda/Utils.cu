#include "../include/cuda/Utils.cuh"

CSRMatrix* uploadCSRToDevice(const CSRMatrix *h_csr) {
    // 1. Crea struttura temporanea su host
    CSRMatrix d_csr;
    d_csr.M = h_csr->M;
    d_csr.N = h_csr->N;
    d_csr.NZ = h_csr->NZ;

    // 2. Alloca array su device e copia i contenuti
    hipMalloc((void**)&d_csr.IRP, (h_csr->M + 1) * sizeof(MatT));
    hipMemcpy(d_csr.IRP, h_csr->IRP, (h_csr->M + 1) * sizeof(MatT), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_csr.JA, h_csr->NZ * sizeof(MatT));
    hipMemcpy(d_csr.JA, h_csr->JA, h_csr->NZ * sizeof(MatT), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_csr.AS, h_csr->NZ * sizeof(MatVal));
    hipMemcpy(d_csr.AS, h_csr->AS, h_csr->NZ * sizeof(MatVal), hipMemcpyHostToDevice);

    // 3. Alloca CSRMatrix su device
    CSRMatrix *d_csr_ptr;
    hipMalloc((void**)&d_csr_ptr, sizeof(CSRMatrix));

    // 4. Copia la struttura su device
    hipMemcpy(d_csr_ptr, &d_csr, sizeof(CSRMatrix), hipMemcpyHostToDevice);

    return d_csr_ptr;
}

void freeCSRDevice(CSRMatrix *d_csr_ptr) {
    CSRMatrix h_temp;
    hipMemcpy(&h_temp, d_csr_ptr, sizeof(CSRMatrix), hipMemcpyDeviceToHost);

    hipFree(h_temp.IRP);
    hipFree(h_temp.JA);
    hipFree(h_temp.AS);
    hipFree(d_csr_ptr);
}