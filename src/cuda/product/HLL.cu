#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "../../include/cuda/Serial.cuh"
#include "../../include/cuda/HLL.cuh"
#include "../../include/createVector.h"
#include "../../include/cuda/Utils.cuh"

#define WARP_SIZE 32 // Impostato a 32 rispetto al server di dipartimento con sopra montata una Quadro RTX 5000
#define BLOCK_SIZE 1024 // Il server ha a disposizione al massimo 1024 thread attivi contemporaneamente
#define MAX_NZ_PER_BLOCK 1024

// Funzione che carica la matrice HLL sulla GPU e la restituisce come puntatore
HLLMatrix* uploadHLLToDevice(HLLMatrix *hll) {
    HLLMatrix *d_hll;

    // Allocazione memoria per la struttura HLLMatrix sulla GPU
    hipMalloc((void**)&d_hll, sizeof(HLLMatrix));

    // Copia della struttura HLLMatrix dalla memoria host alla memoria device
    hipMemcpy(d_hll, hll, sizeof(HLLMatrix), hipMemcpyHostToDevice);

    // Allocazione della memoria per i blocchi ELLPACKMatrix sulla GPU
    hipMalloc((void**)&(d_hll->blocks), hll->numBlocks * sizeof(ELLPACKMatrix*));

    // Ciclo per copiare ogni blocco ELLPACKMatrix sulla GPU
    for (int blockIdx = 0; blockIdx < hll->numBlocks; blockIdx++) {
        ELLPACKMatrix *block = hll->blocks[blockIdx];
        ELLPACKMatrix *d_block;

        // Allocazione memoria per ogni blocco ELLPACKMatrix sulla GPU
        hipMalloc((void**)&d_block, sizeof(ELLPACKMatrix));

        // Copia della struttura ELLPACKMatrix dalla memoria host alla memoria device
        hipMemcpy(d_block, block, sizeof(ELLPACKMatrix), hipMemcpyHostToDevice);

        // Allocazione memoria per i dati del blocco (JA e AS)
        hipMalloc((void**)&(d_block->JA), block->M * block->MAXNZ * sizeof(MatT));
        hipMalloc((void**)&(d_block->AS), block->M * block->MAXNZ * sizeof(MatVal));

        // Copia dei dati di ciascun blocco (JA e AS) dalla memoria host alla memoria device
        for (MatT row = 0; row < block->M; row++) {
            hipMemcpy(d_block->JA[row], block->JA[row], block->MAXNZ * sizeof(MatT), hipMemcpyHostToDevice);
            hipMemcpy(d_block->AS[row], block->AS[row], block->MAXNZ * sizeof(MatVal), hipMemcpyHostToDevice);
        }

        // Aggiornamento dell'array di blocchi nella memoria del dispositivo
        hipMemcpy(&(d_hll->blocks[blockIdx]), &d_block, sizeof(ELLPACKMatrix*), hipMemcpyHostToDevice);
    }

    // Restituzione del puntatore alla matrice HLLMatrix sulla GPU
    return d_hll;
}


__global__ void ellpack_CUDA_product_kernel(ELLPACKMatrix *d_block, MatVal *d_vector, MatVal *d_result) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < d_block->M) {
        MatT *JA = d_block->JA[row];
        MatVal *AS = d_block->AS[row];

        // Moltiplicazione della matrice per il vettore, con somma atomica
        for (int i = 0; i < d_block->MAXNZ; i++) {
            if (JA[i] >= 0) {
                atomicAdd(&d_result[row], AS[i] * d_vector[JA[i]]);
            }
        }
    }
}


ResultVector *hll_CUDA_product(HLLMatrix *h_hll, ResultVector *serial_vector) {
    if (!h_hll || !serial_vector) {
        perror("hll_CUDA_product: NULL pointer detected");
        return NULL;
    }

    // Creazione del vettore risultato sulla GPU
    ResultVector *h_result_vector = create_result_vector(h_hll->M);  // Crea il risultato host
    ResultVector *d_result_vector = uploadResultVectorToDevice(h_result_vector);  // Copia sulla GPU

    // Creazione del vettore di input sulla GPU
    MatVal *h_vector = create_vector(h_hll->N);

    MatVal* d_vector;
    hipMalloc(&d_vector, h_hll->N * sizeof(MatVal));
    hipMemcpy(d_vector, h_vector, h_hll->N * sizeof(MatVal), hipMemcpyHostToDevice);
    hipMemAdvise(d_vector, h_hll->N * sizeof(MatVal), hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(d_vector, h_hll->N * sizeof(MatVal), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);

    // Ciclo sui blocchi HLL
    for (int blockIdx = 0; blockIdx < h_hll->numBlocks; blockIdx++) {
        ELLPACKMatrix *d_block;
        hipMemcpy(&d_block, &h_hll->blocks[blockIdx], sizeof(ELLPACKMatrix*), hipMemcpyHostToDevice);

        // Definizione della griglia di thread
        dim3 threadsPerBlock(256); // Numero di thread per blocco
        dim3 numBlocks((d_block->M + threadsPerBlock.x - 1) / threadsPerBlock.x);

        // Lancio del kernel CUDA
        ellpack_CUDA_product_kernel<<<numBlocks, threadsPerBlock>>>(d_block, d_vector, d_result_vector->val);

        // Controllo errori CUDA
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(err));
        }
    }

    // Copia dei risultati dalla GPU alla memoria host
    hipMemcpy(h_result_vector->val, d_result_vector->val, h_hll->M * sizeof(MatVal), hipMemcpyDeviceToHost);

    // Pulizia della memoria GPU
    hipFree(d_vector);
    hipFree(d_result_vector);

    // Restituisce il vettore risultato
    return h_result_vector;
}


